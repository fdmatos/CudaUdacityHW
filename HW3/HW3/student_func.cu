#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"
#include <fstream>
#include <iostream>
using namespace std;

__global__ void reduce_Minimum(float* d_in,
	int d_inSize,
	int numberOfBlocks,
	float* d_out);

__global__ void reduce_Maximum(float* d_in,
	int d_inSize,
	int numberOfBlocks,
	float* d_out);

__global__ void histogram_SeparateBuckets(const float* const d_in, int* d_threadBucketMatrix,
	int numberOfElements, int elementsPerThread, int pitch,
	float lumMin, float lumRange, int numBins);

__global__ void reduce_SumBuckets(int* d_in, int* d_out, int elementsToProcess);

__global__ void exclusiveScan(unsigned int* d_intermediate, int elementsToProcess);

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

	int numberOfPixels = numRows*numCols;
	int threadsPerBlock = 1024;
	int numberOfBlocks = (numberOfPixels / threadsPerBlock) + 1;

	float* d_blockMinimum;
	float* d_blockMaximum;
	float* d_in;
	checkCudaErrors(hipMalloc(&d_in, numberOfPixels*sizeof(float)));
	checkCudaErrors(hipMemcpy(d_in, d_logLuminance, numberOfPixels*sizeof(float), hipMemcpyDeviceToDevice));

	
	checkCudaErrors(hipMalloc(&d_blockMinimum, numberOfBlocks*sizeof(float)));
	checkCudaErrors(hipMalloc(&d_blockMaximum, numberOfBlocks*sizeof(float)));

	reduce_Minimum << <numberOfBlocks, threadsPerBlock >> >(d_in,
															numberOfPixels, 
															numberOfBlocks, 
															d_blockMinimum);

	reduce_Maximum << <numberOfBlocks, threadsPerBlock >> >(d_in,
															numberOfPixels,
															numberOfBlocks,
															d_blockMaximum);

	reduce_Minimum << <1, threadsPerBlock >> >(d_blockMinimum,
															numberOfBlocks,
															1,
															d_blockMinimum);

	reduce_Maximum << <1, threadsPerBlock >> >(d_blockMaximum,
															numberOfBlocks,
															1,
															d_blockMaximum);

	float* h_blockMinimum = (float*)malloc(numberOfBlocks*sizeof(float));
	checkCudaErrors(hipMemcpy(h_blockMinimum, d_blockMinimum, numberOfBlocks*sizeof(float), hipMemcpyDeviceToHost));
	float* h_blockMaximum = (float*)malloc(numberOfBlocks*sizeof(float));
	checkCudaErrors(hipMemcpy(h_blockMaximum, d_blockMaximum, numberOfBlocks*sizeof(float), hipMemcpyDeviceToHost));
	min_logLum = h_blockMinimum[0];
	max_logLum = h_blockMaximum[0];
	checkCudaErrors(hipFree(d_blockMinimum));
	checkCudaErrors(hipFree(d_blockMaximum));
	
	free(h_blockMaximum);
	free(h_blockMinimum);

	int* d_threadBucketMatrix;
	size_t pitch;
	int pixelsPerThread = numberOfPixels / threadsPerBlock + 1;
	int width = threadsPerBlock*sizeof(int);
	int length = numBins*sizeof(int);
	
	checkCudaErrors(hipMallocPitch(&d_threadBucketMatrix, &pitch, width, length));
	float lumRange = max_logLum - min_logLum;

	////////quantos pixels foram calculados pela thread
	/*int *d_test;
	int somatest = 0;*//*
	checkCudaErrors(hipMalloc(&d_test, threadsPerBlock*sizeof(int)));*/

	histogram_SeparateBuckets << <1, threadsPerBlock >> >(d_logLuminance, d_threadBucketMatrix, numberOfPixels,
		pixelsPerThread, pitch, min_logLum, lumRange, numBins);

	/*int* h_test = (int*)malloc(threadsPerBlock * sizeof(int));
	checkCudaErrors(hipMemcpy(h_test, d_test, threadsPerBlock*sizeof(int), hipMemcpyDeviceToHost));
	ofstream testout("anothertest.txt");
	
	for (int k = 0; k < threadsPerBlock; k++){
		testout << h_test[k];
		somatest+=h_test[k];
	}*/


	int** h_threadBucketMatrix = (int**)malloc(numBins*sizeof(int*));
	for (int k = 0; k < numBins; k++){
		h_threadBucketMatrix[k] = (int*)malloc(threadsPerBlock*sizeof(int));
	}
	for (int k = 0; k < numBins; k++){
		checkCudaErrors(hipMemcpy(h_threadBucketMatrix[k], (int*)((char*)d_threadBucketMatrix + k * pitch), threadsPerBlock*sizeof(int), hipMemcpyDeviceToHost));
	}

	
	/*int summ = 0;
	if (fout.is_open()){
		for (int i = 0; i < numBins; i++){
			for (int k = 0; k < threadsPerBlock; k++){
				fout << (h_threadBucketMatrix[i][k]);
				summ += h_threadBucketMatrix[i][k];
			}
			fout << "\n";
		}
	}*/

	int* h_buckets = (int*)malloc(numBins*sizeof(int));
	int* d_out;
	int* d_bucketValues;
	checkCudaErrors(hipMalloc(&d_bucketValues, threadsPerBlock*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_out, sizeof(int)));
	
	for (int k = 0; k < numBins; k++){
		checkCudaErrors(hipMemcpy(d_bucketValues, h_threadBucketMatrix[k], threadsPerBlock*sizeof(int), hipMemcpyHostToDevice));
		//o numero de elementos a processar nesta fun�ao devia ser o numero de buckets,
		// e nao o numero de threads, que por coincidencia � igual. refactorizar. 
		reduce_SumBuckets << <1, threadsPerBlock >> > (d_bucketValues, d_out, threadsPerBlock);
		checkCudaErrors(hipMemcpy(&h_buckets[k], d_out, sizeof(int), hipMemcpyDeviceToHost));
	
	}

	
	//int sum = 0;
	//if (fout.is_open()){
	//	for (int i = 0; i < numBins; i++){
	//		fout << h_buckets[i];
	//		fout << '-';
	//		sum += h_buckets[i];
	//	}
	//	fout << "\n"; fout << sum;
	//}

	//fazer scan exclusivo de h_buckets

	unsigned int * d_intermediate;
	checkCudaErrors(hipMalloc(&d_intermediate, numBins*sizeof(int)));
	checkCudaErrors(hipMemcpy(d_intermediate, h_buckets, numBins*sizeof(int), hipMemcpyHostToDevice));
	/*int *d_test;
	checkCudaErrors(hipMalloc(&d_test, sizeof(int)));*/
	exclusiveScan << <1, threadsPerBlock >> >(d_intermediate, numBins);
	/*int *h_test = (int*)malloc(sizeof(int));
	checkCudaErrors(hipMemcpy(h_test, d_test, sizeof(int), hipMemcpyDeviceToHost));*/
	checkCudaErrors(hipMemcpy(d_cdf, d_intermediate, numBins * sizeof(int), hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(h_buckets, d_intermediate, numBins*sizeof(int), hipMemcpyDeviceToHost));

	ofstream fout("text.txt");
	for (int k = 0; k < numBins; k++){
		fout << h_buckets[k];
		fout << '-';
	}
	fout << '\n';

//	int test = *h_test;
//	free(h_test);
	free(h_buckets);
	checkCudaErrors(hipFree(d_bucketValues));
	checkCudaErrors(hipFree(d_in));
	checkCudaErrors(hipFree(d_out));
	checkCudaErrors(hipFree(d_threadBucketMatrix));
	for (int i = 0; i < numBins; i++){
		free(h_threadBucketMatrix[i]);
	}
	free(h_threadBucketMatrix);/*
	checkCudaErrors(hipFree(d_test));*/
	checkCudaErrors(hipFree(d_intermediate));
	return;


}


__global__ void exclusiveScan(unsigned int* d_intermediate, int elementsToProcess){
	
	int threadX = threadIdx.x;
	int tid = threadX + blockIdx.x * blockDim.x;
	if (tid > elementsToProcess){
		return;
	}
	
	
	//fase de reduce. dintermediate tem que entrar aqui ja igual a d_in. 
	for (int s = 1, mod = 2; s <= elementsToProcess / 2; s = s * 2, mod = mod * 2){
		if ((threadX + 1) % mod == 0){
			d_intermediate[tid] = d_intermediate[tid] + d_intermediate[tid - s];
		}
		__syncthreads();
	}

	if (threadIdx.x == 1023){
		d_intermediate[1023] = 0;
	}

	__syncthreads();
	//fase de downsweep
	int auxiliary;
	for (int s = elementsToProcess / 2, mod = elementsToProcess; s > 0; s = s / 2, mod = mod / 2){
		if ((threadX + 1) % mod == 0){
			auxiliary = d_intermediate[tid - s];
			d_intermediate[tid - s] = d_intermediate[tid];
			d_intermediate[tid] = d_intermediate[tid] + auxiliary;
		}
		__syncthreads();
	}

	//if (threadIdx.x == 1023){
	//	*d_out = d_intermediate[1023];
	//}
	
}

//Esta fun��o d� asneira se o numero de buckets (o numero de elementos a processar) for maior do que
//o numero de threads definidas por bloco (1024). refactorizar. 
__global__ void reduce_SumBuckets(int* d_in, int* d_out, int elementsToProcess){
	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid > elementsToProcess){
		return;	
	}

	for (int s = elementsToProcess/2; s > 0; s = s / 2){
		if (threadIdx.x < s){
			d_in[tid] = d_in[tid] + d_in[tid + s];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0){
		*d_out = d_in[0];
	}

}

__global__ void histogram_SeparateBuckets(const float* const d_in, int* d_threadBucketMatrix, 
											int numberOfElements, int elementsPerThread, int pitch,
											float lumMin, float lumRange, int numBins){
	

	int pixelToRead;
	int threadX = threadIdx.x;
	int* gridAddress;

	for (int i = 0; i < numBins; i++){
		gridAddress = (int*)((char*)d_threadBucketMatrix + i * pitch) + threadX;
		*gridAddress = 0;
	}
	unsigned int bucket;
	float pixelValue;
//	int test = 0;
	for (int i = 0; i < elementsPerThread; i++){
		pixelToRead = threadX * elementsPerThread + i;
		if (pixelToRead >= numberOfElements){
			//d_test[threadX] = i;
			return;
		}
		pixelValue = d_in[pixelToRead];
		//bucket = floor((((float)(pixelValue - lumMin) / lumRange) * numBins));
		/////
		bucket = fminf(((unsigned int)(numBins - 1)), (unsigned int)((pixelValue - lumMin) / lumRange * numBins));

		/*if (bucket > 1023){
			bucket = 1023;
		}*/
		gridAddress = (int*)((char*)d_threadBucketMatrix + bucket * pitch) + threadX;
		*gridAddress = *gridAddress + 1;
		//test = i;
	}
	//d_test[threadX] = test;
	return;
}



__global__ void reduce_Minimum(float* d_in,
	int d_inSize,
	int numberOfBlocks,
	float* d_out)
{
	int threadsPerBlock = blockDim.x;
	int thisBlockId = blockIdx.x;
	int threadIndex = threadIdx.x;
	int vectorX = thisBlockId * threadsPerBlock + threadIndex;
	if (vectorX > d_inSize){
		return;
	}

	int elementsToProcess;
	if (thisBlockId + 1 == numberOfBlocks){
		elementsToProcess = d_inSize % threadsPerBlock;
	}
	else{
		elementsToProcess = threadsPerBlock;
	}

	int odd = 0;
	for (int s = elementsToProcess / 2; s > 0; s = s / 2){
		if (threadIndex < s){
			odd = elementsToProcess % 2;
			d_in[vectorX] = fminf(d_in[vectorX], d_in[vectorX + s + odd]);
			elementsToProcess -= s;
			if (s % 2 && elementsToProcess == 2){
				s = 2;
			}

		}
		__syncthreads();
	}

	if (threadIndex == 0){
		d_out[thisBlockId] = d_in[vectorX];
	}

	return;
}


__global__ void reduce_Maximum(float* d_in,
	int d_inSize,
	int numberOfBlocks,
	float* d_out)
{
	int threadsPerBlock = blockDim.x;
	int thisBlockId = blockIdx.x;
	int threadIndex = threadIdx.x;
	int vectorX = thisBlockId * threadsPerBlock + threadIndex;
	if (vectorX > d_inSize){
		return;
	}
	
	int elementsToProcess;
	if (thisBlockId + 1 == numberOfBlocks){
		elementsToProcess = d_inSize % threadsPerBlock;
	}
	else{
		elementsToProcess = threadsPerBlock;
	}

	int odd = 0;
	for (int s = elementsToProcess / 2; s > 0; s = s/2){
		if (threadIndex < s){
			odd = elementsToProcess % 2;
			d_in[vectorX] = fmaxf(d_in[vectorX], d_in[vectorX + s + odd]);
			elementsToProcess -= s;
			if (s % 2 && elementsToProcess == 2){
				s = 2;
			}
			
		}
		__syncthreads();
	}

	if (threadIndex == 0){
		d_out[thisBlockId] = d_in[vectorX];
	}

	return;
}


